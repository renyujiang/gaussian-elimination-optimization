#include "hip/hip_runtime.h"
﻿///////////code can run

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>


typedef double data_t;

#define L 1024
#define W 1024
#define error 0.05






void initializeArray(data_t** array, int seed)
{
	srand(seed);
	int i, j;
	if (seed == 0)
	{
		for (i = 0; i < L; i++)
		{
			for (j = 0; j < W + 1; j++)
			{
				array[i][j] = 0;
			}
		}
		return;
	}

	for (i = 0; i < L; i++)
	{
		for (j = 0; j < W + 1; j++)
		{
			int sign = rand() % 2 ? -1 : 1;
			double rand_float = (float)rand() / (float)RAND_MAX * sign;
			array[i][j] = rand_float;
		}
	}
	return;
}




//gaussian_elimination_kernel
__global__ void pivot_and_swap(data_t* a, int n, int k) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;

	if (i == k) {
		int maxIndex = i;
		data_t maxValue = fabs(a[i * (n + 1) + k]);

		// find the largest
		for (int j = i + 1; j < n; j++) {
			data_t currentValue = fabs(a[j * (n + 1) + k]);
			if (currentValue > maxValue) {
				maxIndex = j;
				maxValue = currentValue;
			}
		}

		// swap lines
		if (maxIndex != i) {
			for (int j = k; j <= n; j++) {
				data_t temp = a[i * (n + 1) + j];
				a[i * (n + 1) + j] = a[maxIndex * (n + 1) + j];
				a[maxIndex * (n + 1) + j] = temp;
			}
		}
	}
}

__global__ void elimination(data_t* a, int n, int k) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;

	if (i > k) {
		data_t factor = a[i * (n + 1) + k] / a[k * (n + 1) + k];
		for (int j = k + 1; j <= n; j++) {
			a[i * (n + 1) + j] -= factor * a[k * (n + 1) + j];
		}
	}
}



data_t* gaussian_elimination(data_t* a, int n) {
	data_t* d_a;
	int size = n * (n + 1) * sizeof(data_t);
	hipMalloc((void**)&d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	dim3 blockSize(64);
	dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

	for (int k = 0; k < n - 1; k++) {
		pivot_and_swap << <gridSize, blockSize >> > (d_a, n, k);
		hipDeviceSynchronize();
		elimination << <gridSize, blockSize >> > (d_a, n, k);
		hipDeviceSynchronize();
	}
	hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
	hipFree(d_a);

	data_t* x = (data_t*)malloc(n * sizeof(data_t));
	x[n - 1] = a[n * (n + 1) - 1] / a[n * (n + 1) - 2];
	for (int i = n - 2; i >= 0; i--) {
		data_t s = 0;
		for (int j = i + 1; j < n; j++) {
			s += a[i * (n + 1) + j] * x[j];
		}
		x[i] = (a[i * (n + 1) + n] - s) / a[i * (n + 1) + i];
	}
	return x;

}


//Base Cpu
data_t* gaussian_elimination_base(data_t** a, int n) {
	data_t s, p;
	int i, j, k;
	data_t* x = (data_t*)malloc((L) * sizeof(data_t));
	for (k = 0; k <= n - 1; k++)
	{
		for (i = k + 1; i < n; i++)
		{
			p = a[i][k] / a[k][k];
			for (j = k; j <= n; j++)
			{
				a[i][j] = a[i][j] - (p * a[k][j]);
			}
		}
	}
	x[n - 1] = a[n - 1][n] / a[n - 1][n - 1];
	for (i = n - 2; i >= 0; i--)
	{
		s = 0;
		for (j = i + 1; j < n; j++)
		{
			s += (a[i][j] * x[j]);
		}
		x[i] = (a[i][n] - s) / a[i][i]; // Move this line outside the inner loop
	}
	return x;
}


//ckeck input matrix
void print_matrix(data_t** a, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			printf("%f\t", a[i][j]);
		}
		printf("\n");
	}
}


data_t check_result(data_t* matrix1, data_t* matrix2) {

	//int len1 = sizeof(matrix1) / sizeof(matrix1[0]);
	//int len2 = sizeof(matrix2) / sizeof(matrix2[0]);
	int len1 = L;
	int len2 = L;
	if (len1 != len2) {
		printf("Unable to compare, the two matrixes are not the same size");
		return 0;
	}
	int i;
	data_t real_error, max_error = 0.0;
	for (i = 0; i < len1; i++) {
		real_error = fabs(matrix1[i] - matrix2[i]) / matrix1[i];
		if (real_error > error) {
			printf("The two matrixes' error is larger than the error rate");
			return real_error;
		}
		if (real_error > max_error) {
			max_error = real_error;
		}
	}
	return max_error;
}




int main() {
	data_t** a = (data_t**)
		malloc(L * sizeof(data_t*));
	for (int i = 0; i < L; i++) {
		a[i] = (data_t*)malloc((W + 1) * sizeof(data_t));
	}

	initializeArray(a, 1);


	//put a into 1-D array linearized_a
	data_t* linearized_a = (data_t*)malloc(L * (W + 1) * sizeof(data_t));
	for (int i = 0; i < L; i++) {
		for (int j = 0; j < W + 1; j++) {
			linearized_a[i * (W + 1) + j] = a[i][j];
		}
	}

	//put a into 2-D array a_base
	data_t** a_base = (data_t**)malloc(L * sizeof(data_t*));
	for (int i = 0; i < L; i++) {
		a_base[i] = (data_t*)malloc((W + 1) * sizeof(data_t));
	}
	for (int i = 0; i < L; i++) {
		for (int j = 0; j < W + 1; j++) {
			a_base[i][j] = a[i][j];
		}
	}


	//Result base
	clock_t start_base = clock();
	data_t* x_base = gaussian_elimination_base(a_base, L);
	clock_t end_base = clock();
	double elapsed_time_base = (double)(end_base - start_base) * 1000.0 / CLOCKS_PER_SEC;


	//Result in GPU
	clock_t start = clock();
	data_t* x = gaussian_elimination(linearized_a, L);
	clock_t end = clock();
	double elapsed_time = (double)(end - start) * 1000.0 / CLOCKS_PER_SEC;

	// Compare results and calculate max error
	data_t max_error = check_result(x_base, x);

	// Output the results 

	//print_matrix(a, L, W + 1);
	printf("Base Result:\n");
	for (int i = 0; i < L; i++) {
		printf("x_base[%d] = %f\n", i, x_base[i]);
	}


	printf("Result:\n");
	for (int i = 0; i < L; i++) {
		printf("x[%d] = %f\n", i, x[i]);
	}

	printf("\nBase Elapsed Time: %f ms\n", elapsed_time_base);
	printf("\nElapsed Time: %f ms\n", elapsed_time);
	printf("\nMax Error: %f\n", max_error);
	// Free the allocated memory
	for (int i = 0; i < L; i++) {
		free(a[i]);
	}


	free(a);
	free(linearized_a);
	free(x);

	return 0;
}